#include "hip/hip_runtime.h"
#include "src/kernels/layer_norm.h"
#include "src/core/check_cuda.h"
#include "src/kernels/_device_.cuh"

namespace cudaTransformer {

template <typename T>
__device__ T warpReduceSum(T val)
{
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);  
    }
    return val;
}

template <typename T>
__global__ void layerNormKernel(const Tensor3D_Kernel<T> input, 
                                Tensor3D_Kernel<T> output,
                                const Tensor2D_Kernel<T> alpha,
                                const Tensor2D_Kernel<T> beta,
                                T epsilon) {

    const int block_size = 1024;

    int hidden_size = input.row_size;

    int row = blockIdx.x;

    int col = threadIdx.y;

    int batch = blockIdx.z;

    T val = (T)0;

    __shared__ T buffer_mean_sum[block_size];
    __shared__ T buffer_std_sum[block_size];

    __shared__ T mean;
    __shared__ T std;

    if (threadIdx.y < hidden_size) {
        val = getValTensor(input, batch, row, col);
        buffer_mean_sum[threadIdx.y] = val;
        buffer_std_sum[threadIdx.y] = val;
    } else {
        buffer_mean_sum[threadIdx.y] = (T)0;
        buffer_std_sum[threadIdx.y] = (T)0;
    }

    __syncthreads();

    for (int offset = block_size >> 1; offset >= warpSize; offset >>= 1) {
        if (threadIdx.y < offset) {
            buffer_mean_sum[threadIdx.y] += buffer_mean_sum[threadIdx.y + offset];
        }
        __syncthreads();
    }

    T reg_mean_sum = buffer_mean_sum[threadIdx.y];

    reg_mean_sum = warpReduceSum(reg_mean_sum);

    if (threadIdx.y == 0) mean = __fdividef(reg_mean_sum, hidden_size);

    __syncthreads();

    if (threadIdx.y < hidden_size) {
        buffer_std_sum[threadIdx.y] = powf(buffer_std_sum[threadIdx.y] - mean, 2.0f);
    } 

    __syncthreads();

    for (int offset = block_size >> 1; offset >= warpSize; offset >>= 1) {
        
        if (threadIdx.y < offset) {
            buffer_std_sum[threadIdx.y] += buffer_std_sum[threadIdx.y + offset];
        }
        __syncthreads();
    }

    T reg_std_sum = buffer_std_sum[threadIdx.y];

    reg_std_sum = warpReduceSum(reg_std_sum);

    if (threadIdx.y == 0) std = sqrtf(__fdividef(reg_std_sum, hidden_size));

    __syncthreads();

    // element-wise

    if (threadIdx.y < hidden_size) {

        T x = getValTensor(input, batch, row, col);

        // x = alpha.data[threadIdx.y] * (x - mean) / (std + epsilon) + beta.data[threadIdx.y];
        x = __fdividef(alpha.data[threadIdx.y] * (x - mean), std + epsilon) + beta.data[threadIdx.y];

        setValTensor(output, batch, row, col, x);

    }
    

}

template <typename T>
void layerNorm(const Tensor<T>* input, 
                Tensor<T>* output,
                const Tensor<T>* alpha, 
                const Tensor<T>* beta, 
                T epsilon) {
                    
    Tensor3D_Kernel<T> _input(*input);
    Tensor3D_Kernel<T> _output(*output);
    Tensor2D_Kernel<T> _alpha(*alpha);
    Tensor2D_Kernel<T> _beta(*beta);

    int dim_x = _output.col_size;
    int dim_y = _output.row_size;
    int dim_z = _output.batch_size;

    const int block_size = 1024;
    // support hidden_size = 768, 1024

    dim3 block(1, block_size, 1);
    dim3 grid(dim_x / block.x , (dim_y - 1) / block.y + 1, dim_z / block.z);
    layerNormKernel<T><<<grid, block>>>(_input, _output, _alpha, _beta, epsilon);
    CHECK(hipGetLastError());
    // hipDeviceSynchronize();
}
template void layerNorm<float>(const Tensor<float>* input, 
                                Tensor<float>* output,
                                const Tensor<float>* alpha, 
                                const Tensor<float>* beta, 
                                float epsilon);

}